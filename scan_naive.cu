
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

#define SIZE 3000
#define BLOCK_SIZE 1024
__global__ void prefix_sum(int *in, int *out) {
  int cur_index = blockIdx.x * blockDim.x + threadIdx.x;  // allocate memory
  int res = 0;
  for (int i = 0; i <= cur_index; i++){
      res += in[i];
  }
  out[cur_index] = res;
}

int main() {
   // allocate memory
   int *input, *output;
   hipMallocManaged(&input, sizeof(int) * SIZE);
   hipMallocManaged(&output, sizeof(int) * SIZE);

   // initialize inputs
   for (int i = 0; i < SIZE; i++) {
       input[i] = 1;
   }
   int block_num;
   if (SIZE % BLOCK_SIZE != 0){
       block_num = SIZE/BLOCK_SIZE + 1;
   }
   clock_t a = clock();
   prefix_sum<<<block_num, BLOCK_SIZE>>>(input, output);
   hipDeviceSynchronize();
   clock_t b = clock() - a;
   printf("time: %f ", (float) b/CLOCKS_PER_SEC);
   for (int i = 0; i < SIZE; i++) {
       printf("%d ", output[i]);
   }
   printf("\n");

   printf("%s\n", hipGetErrorString(hipGetLastError()));
   hipFree(input);
   hipFree(output);
   return 0;
}