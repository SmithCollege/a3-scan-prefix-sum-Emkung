
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

#define SIZE 128
#define BLOCK_SIZE 128
__global__ void prefix_sum(int *in, int *out) {
  int cur_index = blockIdx.x * blockDim.x + threadIdx.x;  // allocate memory
  int res = 0;
  __syncthreads();
  for (int i = 0; i <= cur_index; i++){
      res += in[i];
  }
  out[cur_index] = res;
}

int main() {
   // allocate memory
   int *input, *output;
   hipMallocManaged(&input, sizeof(int) * SIZE);
   hipMallocManaged(&output, sizeof(int) * SIZE);

   // initialize inputs
   for (int i = 0; i < SIZE; i++) {
       input[i] = 1;
   }

   clock_t a = clock();
   prefix_sum<<<SIZE/BLOCK_SIZE, BLOCK_SIZE>>>(input, output);
   hipDeviceSynchronize();
   clock_t b = clock() - a;
   printf("time: %f ", (float) b/CLOCKS_PER_SEC);
   for (int i = 0; i < SIZE; i++) {
       printf("%d ", output[i]);
   }
   printf("\n");

   printf("%s\n", hipGetErrorString(hipGetLastError()));
   hipFree(input);
   hipFree(output);
   return 0;
}