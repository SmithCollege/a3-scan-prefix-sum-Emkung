
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <math.h>

#define SIZE 3000
#define BLOCK_SIZE 1024
__global__ void double_recursion_scan(int *in, int *out) {
  int cur_index = blockIdx.x * blockDim.x + threadIdx.x;  // allocate memory
  for (int stride = 1; stride/2 <= SIZE; stride *= 2  ){
      __syncthreads();
      if (cur_index < stride){
      	 out[cur_index] = in[cur_index];
      }else{
         out[cur_index] = in[cur_index] + in[cur_index - stride];
         int* temp = out;
         out = in;
	 in = temp;
       }
   }
}

int main() {
   // allocate memory
   int *input, *output;
   hipMallocManaged(&input, sizeof(int) * SIZE);
   hipMallocManaged(&output, sizeof(int) * SIZE);

   // initialize inputs
   for (int i = 0; i < SIZE; i++) {
       input[i] = 1;
   }

   int block_num;
   if (SIZE % BLOCK_SIZE != 0){
       block_num = SIZE/BLOCK_SIZE + 1;
   }
   clock_t a = clock();
   double_recursion_scan<<<block_num, BLOCK_SIZE>>>(input, output);
   hipDeviceSynchronize();
   clock_t b = clock() - a;
   printf("time: %f ", (float) b/CLOCKS_PER_SEC);
   for (int i = 0; i < SIZE; i++) {
       printf("%d ", output[i]);
   }
   printf("\n");

   printf("%s\n", hipGetErrorString(hipGetLastError()));
   hipFree(input);
   hipFree(output);
   return 0;
}